#include "hip/hip_runtime.h"
#include <limits>
#include <hipcub/hipcub.hpp>
#include <c10/cuda/CUDACachingAllocator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/for_each.h>

#include "epic_ops/reduce.h"
#include "epic_ops/utils/thrust_allocator.h"

namespace epic_ops::reduce {

template <typename scalar_t, int N>
struct array_t {
  scalar_t data[N];

  inline void fill(scalar_t x) {
    #pragma unroll
    for (int i = 0; i < N; i++) {
      data[i] = x;
    }
  }
};

template <typename scalar_t, int N>
struct array_sum_op {
  using value_type = array_t<scalar_t, N>;

  __forceinline__
  __host__ __device__
  value_type operator() (const value_type& a, const value_type& b) const {
    value_type res;
    #pragma unroll
    for (int i = 0; i < N; i++) {
      res.data[i] = a.data[i] + b.data[i];
    }
    return res;
  }
};

template <typename scalar_t, int N>
struct array_min_op {
  using value_type = array_t<scalar_t, N>;

  __forceinline__
  __host__ __device__
  value_type operator() (const value_type& a, const value_type& b) const {
    value_type res;
    #pragma unroll
    for (int i = 0; i < N; i++) {
      res.data[i] = a.data[i] < b.data[i] ? a.data[i] : b.data[i];
    }
    return res;
  }
};

template <typename scalar_t, int N>
struct array_max_op {
  using value_type = array_t<scalar_t, N>;

  __forceinline__
  __host__ __device__
  value_type operator() (const value_type& a, const value_type& b) const {
    value_type res;
    #pragma unroll
    for (int i = 0; i < N; i++) {
      res.data[i] = a.data[i] > b.data[i] ? a.data[i] : b.data[i];
    }
    return res;
  }
};

template <typename scalar_t, typename index_t, int num_channels>
void segmented_reduce_cuda_impl(
    at::Tensor& output,
    const at::Tensor& values,
    const at::Tensor& segment_offsets_begin,
    const at::Tensor& segment_offsets_end,
    int64_t mode) {
  auto num_segments = segment_offsets_begin.size(0);

  auto output_ptr = reinterpret_cast<array_t<scalar_t, num_channels>*>(
      output.data_ptr<scalar_t>());
  auto values_ptr = reinterpret_cast<array_t<scalar_t, num_channels>*>(
      values.data<scalar_t>());
  auto segment_offsets_begin_ptr = segment_offsets_begin.data<index_t>();
  auto segment_offsets_end_ptr = segment_offsets_end.data<index_t>();

  void* d_temp_storage = nullptr;
  size_t num_temp_storage_bytes = 0;
  array_t<scalar_t, num_channels> initial_value;

  if (mode == 0) {  // sum
    initial_value.fill(0);
    hipcub::DeviceSegmentedReduce::Reduce(
        d_temp_storage, num_temp_storage_bytes,
        values_ptr, output_ptr, num_segments,
        segment_offsets_begin_ptr, segment_offsets_end_ptr,
        array_sum_op<scalar_t, num_channels>(),
        initial_value);
  } else if (mode == 1) { // min
    initial_value.fill(std::numeric_limits<scalar_t>::max());
    hipcub::DeviceSegmentedReduce::Reduce(
        d_temp_storage, num_temp_storage_bytes,
        values_ptr, output_ptr, num_segments,
        segment_offsets_begin_ptr, segment_offsets_end_ptr,
        array_min_op<scalar_t, num_channels>(),
        initial_value);
  } else {  // max
    initial_value.fill(std::numeric_limits<scalar_t>::min());
    hipcub::DeviceSegmentedReduce::Reduce(
        d_temp_storage, num_temp_storage_bytes,
        values_ptr, output_ptr, num_segments,
        segment_offsets_begin_ptr, segment_offsets_end_ptr,
        array_max_op<scalar_t, num_channels>(),
        initial_value);
  }

  d_temp_storage = c10::cuda::CUDACachingAllocator::raw_alloc(num_temp_storage_bytes);

  if (mode == 0) {  // sum
    hipcub::DeviceSegmentedReduce::Reduce(
        d_temp_storage, num_temp_storage_bytes,
        values_ptr, output_ptr, num_segments,
        segment_offsets_begin_ptr, segment_offsets_end_ptr,
        array_sum_op<scalar_t, num_channels>(),
        initial_value);
  } else if (mode == 1) { // min
    hipcub::DeviceSegmentedReduce::Reduce(
        d_temp_storage, num_temp_storage_bytes,
        values_ptr, output_ptr, num_segments,
        segment_offsets_begin_ptr, segment_offsets_end_ptr,
        array_min_op<scalar_t, num_channels>(),
        initial_value);
  } else {  // max
    hipcub::DeviceSegmentedReduce::Reduce(
        d_temp_storage, num_temp_storage_bytes,
        values_ptr, output_ptr, num_segments,
        segment_offsets_begin_ptr, segment_offsets_end_ptr,
        array_max_op<scalar_t, num_channels>(),
        initial_value);
  }

  c10::cuda::CUDACachingAllocator::raw_delete(d_temp_storage);
}

at::Tensor segmented_reduce_cuda(
    const at::Tensor& values,
    const at::Tensor& segment_offsets_begin,
    const at::Tensor& segment_offsets_end,
    int64_t mode) {
  TORCH_CHECK(values.is_cuda(), "values must be a CUDA tensor");
  TORCH_CHECK(segment_offsets_begin.is_cuda(), "segment_offsets_begin must be a CUDA tensor");
  TORCH_CHECK(segment_offsets_end.is_cuda(), "segment_offsets_end must be a CUDA tensor");

  TORCH_CHECK(values.dim() == 2, "values must be a 2D tensor");
  TORCH_CHECK(segment_offsets_begin.dim() == 1, "segment_offsets_begin must be a 1D tensor");
  TORCH_CHECK(segment_offsets_end.dim() == 1, "segment_offsets_end must be a 1D tensor");
  TORCH_CHECK(segment_offsets_begin.size(0) == segment_offsets_end.size(0),
              "segment_offsets_begin and segment_offsets_end must have the same size");

  TORCH_CHECK(values.is_contiguous(), "values must be contiguous");
  TORCH_CHECK(segment_offsets_begin.is_contiguous(), "segment_offsets_begin must be contiguous");
  TORCH_CHECK(segment_offsets_end.is_contiguous(), "segment_offsets_end must be contiguous");

  TORCH_CHECK(0 <= mode && mode <= 2, "mode must be in [0, 2]");

  auto num_segments = segment_offsets_begin.size(0);
  auto num_channels = values.size(1);
  TORCH_CHECK(1 <= num_channels && num_channels <= 4, "num_channels must be in [1, 4]");

  auto output = at::empty({num_segments, num_channels}, values.options());

  AT_DISPATCH_FLOATING_TYPES(values.type(), "segmented_reduce_cuda", [&] {
    if (segment_offsets_begin.scalar_type() == at::kInt) {
      if (num_channels == 1) {
        segmented_reduce_cuda_impl<scalar_t, int32_t, 1>(
            output, values, segment_offsets_begin, segment_offsets_end, mode);
      } else if (num_channels == 2) {
        segmented_reduce_cuda_impl<scalar_t, int32_t, 2>(
            output, values, segment_offsets_begin, segment_offsets_end, mode);
      } else if (num_channels == 3) {
        segmented_reduce_cuda_impl<scalar_t, int32_t, 3>(
            output, values, segment_offsets_begin, segment_offsets_end, mode);
      } else if (num_channels == 4) {
        segmented_reduce_cuda_impl<scalar_t, int32_t, 4>(
            output, values, segment_offsets_begin, segment_offsets_end, mode);
      } else {
        AT_ERROR("Unsupported number of channels");
      }
    } else if (segment_offsets_begin.scalar_type() == at::kLong) {
      if (num_channels == 1) {
        segmented_reduce_cuda_impl<scalar_t, int64_t, 1>(
            output, values, segment_offsets_begin, segment_offsets_end, mode);
      } else if (num_channels == 2) {
        segmented_reduce_cuda_impl<scalar_t, int64_t, 2>(
            output, values, segment_offsets_begin, segment_offsets_end, mode);
      } else if (num_channels == 3) {
        segmented_reduce_cuda_impl<scalar_t, int64_t, 3>(
            output, values, segment_offsets_begin, segment_offsets_end, mode);
      } else if (num_channels == 4) {
        segmented_reduce_cuda_impl<scalar_t, int64_t, 4>(
            output, values, segment_offsets_begin, segment_offsets_end, mode);
      } else {
        AT_ERROR("Unsupported number of channels");
      }
    } else {
      AT_ERROR("Unsupported type (segmented_reduce_cuda)");
    }
  });

  return output;
}

template <typename scalar_t, typename index_t>
void segmented_maxpool_cuda_impl(
    at::Tensor& output,
    at::Tensor& max_indices,
    const at::Tensor& values,
    const at::Tensor& segment_offsets_begin,
    const at::Tensor& segment_offsets_end) {
  auto stream = at::cuda::getCurrentCUDAStream().stream();
  auto policy = thrust::cuda::par(utils::ThrustAllocator()).on(stream);

  index_t num_segments = segment_offsets_begin.size(0);
  index_t num_channels = values.size(1);

  auto output_ptr = output.data_ptr<scalar_t>();
  auto max_indices_ptr = max_indices.data_ptr<index_t>();
  auto values_ptr = values.data_ptr<scalar_t>();
  auto segment_offsets_begin_ptr = segment_offsets_begin.data_ptr<index_t>();
  auto segment_offsets_end_ptr = segment_offsets_end.data_ptr<index_t>();

  thrust::for_each(
      policy,
      thrust::counting_iterator<index_t>(0),
      thrust::counting_iterator<index_t>(num_segments * num_channels),
      [=] __host__ __device__ (index_t idx) {
        index_t segment_idx = idx / num_channels;
        index_t channel_idx = idx % num_channels;

        auto begin = segment_offsets_begin_ptr[segment_idx];
        auto end = segment_offsets_end_ptr[segment_idx];

        index_t max_idx = -1;
        scalar_t max_value = -std::numeric_limits<scalar_t>::max();

        for (auto i = begin; i < end; i++) {
          auto value = values_ptr[i * num_channels + channel_idx];
          if (value > max_value) {
            max_value = value;
            max_idx = i;
          }
        }

        output_ptr[idx] = max_value;
        max_indices_ptr[idx] = max_idx;
      });
}

std::tuple<at::Tensor, at::Tensor> segmented_maxpool_cuda(
    const at::Tensor& values,
    const at::Tensor& segment_offsets_begin,
    const at::Tensor& segment_offsets_end) {
  TORCH_CHECK(values.is_cuda(), "values must be a CUDA tensor");
  TORCH_CHECK(segment_offsets_begin.is_cuda(), "segment_offsets_begin must be a CUDA tensor");
  TORCH_CHECK(segment_offsets_end.is_cuda(), "segment_offsets_end must be a CUDA tensor");

  TORCH_CHECK(values.dim() == 2, "values must be a 2D tensor");
  TORCH_CHECK(segment_offsets_begin.dim() == 1, "segment_offsets_begin must be a 1D tensor");
  TORCH_CHECK(segment_offsets_end.dim() == 1, "segment_offsets_end must be a 1D tensor");
  TORCH_CHECK(segment_offsets_begin.size(0) == segment_offsets_end.size(0),
              "segment_offsets_begin and segment_offsets_end must have the same size");

  TORCH_CHECK(values.is_contiguous(), "values must be contiguous");
  TORCH_CHECK(segment_offsets_begin.is_contiguous(), "segment_offsets_begin must be contiguous");
  TORCH_CHECK(segment_offsets_end.is_contiguous(), "segment_offsets_end must be contiguous");

  auto num_segments = segment_offsets_begin.size(0);
  auto num_channels = values.size(1);

  auto output = at::empty({num_segments, num_channels}, values.options());
  auto max_indices = at::empty({num_segments, num_channels}, segment_offsets_begin.options());

  AT_DISPATCH_FLOATING_TYPES(values.type(), "segmented_maxpool_cuda", [&] {
    if (segment_offsets_begin.scalar_type() == at::kInt) {
      segmented_maxpool_cuda_impl<scalar_t, int32_t>(
          output, max_indices, values, segment_offsets_begin, segment_offsets_end);
    } else if (segment_offsets_begin.scalar_type() == at::kLong) {
      segmented_maxpool_cuda_impl<scalar_t, int64_t>(
          output, max_indices, values, segment_offsets_begin, segment_offsets_end);
    } else {
      AT_ERROR("Unsupported type (segmented_maxpool_cuda)");
    }
  });

  return {output, max_indices};
}

TORCH_LIBRARY_IMPL(epic_ops, CUDA, m) {
  m.impl(TORCH_SELECTIVE_NAME("epic_ops::segmented_reduce"),
         TORCH_FN(segmented_reduce_cuda));

  m.impl(TORCH_SELECTIVE_NAME("epic_ops::segmented_maxpool"),
         TORCH_FN(segmented_maxpool_cuda));
}

} // namespace epic_ops::reduce
